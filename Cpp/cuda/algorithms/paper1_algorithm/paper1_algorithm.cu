#include "hip/hip_runtime.h"
#include "../../preprocessing.h"  // getting common processes
#include <chrono>

// CUDA
__global__ void copyUpperLower(int* mat, int* upper, int* lower, int numVertices_edgeList) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < numVertices_edgeList && j < numVertices_edgeList) {
        if (i <= j) {
            upper[i * numVertices_edgeList + j] = mat[i * numVertices_edgeList + j];
            lower[j * numVertices_edgeList + i] = mat[i * numVertices_edgeList + j];
        }
    }
}
__global__ void matrixMultiplication(int* product, int* upper, int* lower, int numVertices_edgeList) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;
    for (int k = 0; k < numVertices_edgeList; ++k) {
        sum += upper[i * numVertices_edgeList + k] * lower[k * numVertices_edgeList + j];
    }
    product[i * numVertices_edgeList + j] = sum;
}
__global__ void matrixElementWiseMultiply(int* mat, int* product, int* result, int numVertices_edgeList) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < numVertices_edgeList && j < numVertices_edgeList) {
        result[i * numVertices_edgeList + j] = mat[i * numVertices_edgeList + j] * product[i * numVertices_edgeList + j];
    }
}
__global__ void sumResultMatrix(int* result, int* sum, int numVertices_edgeList) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < numVertices_edgeList && j < numVertices_edgeList) {
        atomicAdd(sum, result[i * numVertices_edgeList + j]);
    }
}
///////////////

int algorithm1_gpu(const std::string& filename, std::vector<std::pair<int, int>>& edges){
//    std::vector<std::pair<int, int>> edgeList = edgeLine_parser(filename);
    std::vector<std::pair<int, int>> edgeList = edges;
    int numVertices_edgeList = getNumberOfVertices(edgeList);

    int sum = 0;

    int size = numVertices_edgeList * numVertices_edgeList * sizeof(int);
    std::size_t arr_size = numVertices_edgeList * numVertices_edgeList;
    int *mat = new int[arr_size]();

    // making adjacency matrix
    for (const auto &edge : edgeList) {
        int vertex1 = edge.first;
        int vertex2 = edge.second;

        // Set the elements to 1 to indicate the presence of an edge
        mat[vertex1 * numVertices_edgeList + vertex2] = 1;
        mat[vertex2 * numVertices_edgeList + vertex1] = 1;
    }
    std::cout << "graph parsed" << std::endl;

    auto start_time = std::chrono::high_resolution_clock::now();

    int *upper = new int[arr_size]();
    int *lower = new int[arr_size]();

    // splitting into upper and lower

    // Allocate device memory for mat, upper, and lower
    int* d_mat;
    int* d_upper;
    int* d_lower;

    hipMalloc((void**)&d_mat, size);
    hipMalloc((void**)&d_upper, size);
    hipMalloc((void**)&d_lower, size);

    // Copy data from host to device
    hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions for the CUDA kernel
    dim3 dimGrid4((numVertices_edgeList + 15) / 16, (numVertices_edgeList + 15) / 16);
    dim3 dimBlock4(16, 16);

    // Launch the CUDA kernel to copy upper and lower triangular elements
    copyUpperLower<<<dimGrid4, dimBlock4>>>(d_mat, d_upper, d_lower, numVertices_edgeList);

    // Copy the results back from device to host if needed
    hipMemcpy(upper, d_upper, size, hipMemcpyDeviceToHost);
    hipMemcpy(lower, d_lower, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_mat);
    hipFree(d_upper);
    hipFree(d_lower);


    int *product = new int[arr_size]();


    // multiply upper and lower
    int *d_product;

    // Allocate memory on the GPU
    hipMalloc((void**)&d_product, size);
    hipMalloc((void**)&d_upper, size);
    hipMalloc((void**)&d_lower, size);

    // Copy data from host to device
    hipMemcpy(d_upper, upper, size, hipMemcpyHostToDevice);
    hipMemcpy(d_lower, lower, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid(numVertices_edgeList, numVertices_edgeList);
    dim3 dimBlock(1, 1); // Adjust the block size as needed

    // Launch the CUDA kernel
    matrixMultiplication<<<dimGrid, dimBlock>>>(d_product, d_upper, d_lower, numVertices_edgeList);

    // Copy the result back to the host
    hipMemcpy(product, d_product, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_product);
    hipFree(d_upper);
    hipFree(d_lower);
    // multiplying upper and lower

    delete[] upper;
    delete[] lower;

    int *result = new int[arr_size]();

    int* d_result;

    hipMalloc((void**)&d_mat, size);
    hipMalloc((void**)&d_product, size);
    hipMalloc((void**)&d_result, size);

    // Copy data from host to device
    hipMemcpy(d_mat, mat, size, hipMemcpyHostToDevice);
    hipMemcpy(d_product, product, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions for the CUDA kernel
    dim3 dimGrid2((numVertices_edgeList + 15) / 16, (numVertices_edgeList + 15) / 16);
    dim3 dimBlock2(16, 16);

    // Launch the CUDA kernel
    matrixElementWiseMultiply<<<dimGrid2, dimBlock2>>>(d_mat, d_product, d_result, numVertices_edgeList);

    // Copy the result back from device to host
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_mat);
    hipFree(d_product);
    hipFree(d_result);

    // element wise multiplication of mat and product


    delete[] mat;
    delete[] product;


    // summing up the matrix


    // Allocate device memory for result and sum
    int* d_sum;

    hipMalloc((void**)&d_result, size);
    hipMalloc((void**)&d_sum, sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_result, result, size, hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &sum, sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions for the CUDA kernel
    dim3 dimGrid3((numVertices_edgeList + 15) / 16, (numVertices_edgeList + 15) / 16);
    dim3 dimBlock3(16, 16);

    // Launch the CUDA kernel to compute the sum
    sumResultMatrix<<<dimGrid3, dimBlock3>>>(d_result, d_sum, numVertices_edgeList);

    // Copy the sum back from device to host
    hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_result);
    hipFree(d_sum);

    delete[] result;
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    long long elapsed_time = duration.count();
    std::cout << "GPU time run (milliseconds): " << elapsed_time << std::endl;
    return sum;
}

int algorithm1_cpu(const std::string& filename, std::vector<std::pair<int, int>>& edges){
//    std::vector<std::pair<int, int>> edgeList = edgeLine_parser(filename);
    std::vector<std::pair<int, int>> edgeList = edges;
    int numVertices_edgeList = getNumberOfVertices(edgeList);

    int sum = 0;

    int size = numVertices_edgeList * numVertices_edgeList * sizeof(int);
    std::size_t arr_size = numVertices_edgeList * numVertices_edgeList;
    int *mat = new int[arr_size]();
    int *upper = new int[arr_size]();
    int *lower = new int[arr_size]();
    int *product = new int[arr_size]();
    int *result = new int[arr_size]();
    // making adjacency matrix
    for (const auto &edge : edgeList) {
        int vertex1 = edge.first;
        int vertex2 = edge.second;

        // Set the elements to 1 to indicate the presence of an edge
        mat[vertex1 * numVertices_edgeList + vertex2] = 1;
        mat[vertex2 * numVertices_edgeList + vertex1] = 1;
    }

    auto start_time = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < numVertices_edgeList; i++) {
        for (int j = 0; j < numVertices_edgeList; j++) {
            if (i <= j) {
                upper[i * numVertices_edgeList + j] = mat[i * numVertices_edgeList + j];
                lower[j * numVertices_edgeList + i] = mat[i * numVertices_edgeList + j];
            }
        }
    }

    for (int i = 0; i < numVertices_edgeList; ++i) {
        for (int j = 0; j < numVertices_edgeList; ++j) {
            for (int k = 0; k < numVertices_edgeList; ++k) {
                product[i * numVertices_edgeList + j] += upper[i * numVertices_edgeList + k] * lower[k * numVertices_edgeList + j];
            }
        }
    }

    delete[] upper;
    delete[] lower;

    for (int i = 0; i < numVertices_edgeList; i++) {
        for (int j = 0; j < numVertices_edgeList; j++) {
            // Calculate the element-wise product and store it in the result matrix
            result[i * numVertices_edgeList + j] = mat[i * numVertices_edgeList + j] * product[i * numVertices_edgeList + j];
        }
    }

    for (int i = 0; i < numVertices_edgeList; i++) {
        for (int j = 0; j < numVertices_edgeList; j++) {
            sum += result[i * numVertices_edgeList + j];
        }
    }

    delete[] result;
    delete[] product;
    delete[] mat;

    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    long long elapsed_time = duration.count();
    std::cout << "GPU time run (milliseconds): " << elapsed_time << std::endl;
    return sum;
}